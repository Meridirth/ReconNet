#include "hip/hip_runtime.h"
#include "Siddon.h"
#include "Projector.h"
#include "cudaMath.h"

#include <stdexcept>
#include <iostream>
#include <sstream>

using namespace std;

__device__ __host__ float3 GetDstForCone(float u, float v,
		const float3& detCenter, const float3& detU, const float3& detV, const Grid grid)
{
	return make_float3(detCenter.x + detU.x * u + detV.x * v,
		detCenter.y + detU.y * u + detV.y * v,
		detCenter.z + detU.z * u + detV.z * v);

}

__global__ void SiddonConeProjectionAbitraryKernel(float* pPrj, const float* pImg,
	const float3* pDetCenter, const float3* pDetU, const float3* pDetV, const float3* pSrc,
	int nview, int nc, const Detector det, const Grid grid)
{
	int iu = blockDim.x * blockIdx.x + threadIdx.x;
	int iview = blockDim.y * blockIdx.y + threadIdx.y;
	int iv = blockDim.z * blockIdx.z + threadIdx.z;

	if (iu >= det.nu || iv >= det.nv || iview >= nview)
	{
		return;
	}

	float u = (iu - det.off_u - (det.nu - 1) / 2.0f) * det.du;
	float v = (iv - det.off_v - (det.nv - 1) / 2.0f) * det.dv;

	float3 src = pSrc[iview];
	float3 dst = GetDstForCone(u, v, pDetCenter[iview], pDetU[iview], pDetV[iview], grid);
	MoveSourceDstNearGrid(src, dst, grid);

	SiddonRayTracing(pPrj + iu * nview * det.nv * nc + iview * det.nv * nc + iv * nc,
				pImg, src, dst, nc, grid);

}

__global__ void SiddonConeBackprojectionAbitraryKernel(float* pImg, const float* pPrj,
		const float3* pDetCenter, const float3* pDetU, const float3* pDetV, const float3* pSrc,
		int nview, int nc, const Detector det, const Grid grid)
{
	int iu = blockDim.x * blockIdx.x + threadIdx.x;
	int iview = blockDim.y * blockIdx.y + threadIdx.y;
	int iv = blockDim.z * blockIdx.z + threadIdx.z;

	if (iu >= det.nu || iv >= det.nv || iview >= nview)
	{
		return;
	}

	float u = (iu - det.off_u - (det.nu - 1) / 2.0f) * det.du;
	float v = (iv - det.off_v - (det.nv - 1) / 2.0f) * det.dv;

	float3 src = pSrc[iview];
	float3 dst = GetDstForCone(u, v, pDetCenter[iview], pDetU[iview], pDetV[iview], grid);
	MoveSourceDstNearGrid(src, dst, grid);

	SiddonRayTracingTransposeAtomicAdd(pImg, pPrj[iu * nview * det.nv * nc + iview * det.nv * nc + iv * nc],
			src, dst, nc, grid);

}

void SiddonCone::ProjectionAbitrary(const float* pcuImg, float* pcuPrj, const float* pcuDetCenter,
		const float* pcuDetU, const float* pcuDetV, const float* pcuSrc)
{
	dim3 threads, blocks;
	GetThreadsForXZ(threads, blocks, nu, nview, nv);

	for (int ib = 0; ib < nBatches; ib++)
	{
		for (int ic = 0; ic < nChannels; ic++)
		{
			SiddonConeProjectionAbitraryKernel<<<blocks, threads, 0, m_stream>>>(
					pcuPrj + ib * nu * nview * nv * nChannels + ic,
					pcuImg + ib * nx * ny * nz * nChannels + ic,
					(const float3*)pcuDetCenter, (const float3*)pcuDetU,
					(const float3*)pcuDetV, (const float3*)pcuSrc,
					nview, nChannels, MakeDetector(nu, nv, du, dv, off_u, off_v),
					MakeGrid(nx, ny, nz, dx, dy, dz, cx, cy, cz));

			hipDeviceSynchronize();
		}
	}

}

void SiddonCone::BackprojectionAbitrary(float* pcuImg, const float* pcuPrj, const float* pcuDetCenter,
			const float* pcuDetU, const float* pcuDetV, const float* pcuSrc)
{
	dim3 threads, blocks;
	GetThreadsForXZ(threads, blocks, nu, nview, nv);

	for (int ib = 0; ib < nBatches; ib++)
	{
		for (int ic = 0; ic < nChannels; ic++)
		{
			SiddonConeBackprojectionAbitraryKernel<<<blocks, threads, 0, m_stream>>>(
					pcuImg + ib * nx * ny * nz * nChannels + ic,
					pcuPrj + ib * nu * nview * nv * nChannels + ic,
					(const float3*)pcuDetCenter, (const float3*)pcuDetU,
					(const float3*)pcuDetV, (const float3*)pcuSrc,
					nview, nChannels, MakeDetector(nu, nv, du, dv, off_u, off_v),
					MakeGrid(nx, ny, nz, dx, dy, dz, cx, cy, cz));

			hipDeviceSynchronize();
		}
	}
}

extern "C" void cSiddonConeProjectionAbitrary(float* prj, const float* img,
		const float* detCenter, const float* detU, const float* detV, const float* src,
		int nBatches, int nChannels,
		int nx, int ny, int nz, float dx, float dy, float dz, float cx, float cy, float cz,
		int nu, int nview, int nv, float du, float dv, float off_u, float off_v)
{
	float* pcuPrj = NULL;
	float* pcuImg = NULL;
	float* pcuDetCenter = NULL;
	float* pcuDetU = NULL;
	float* pcuDetV = NULL;
	float* pcuSrc = NULL;

	try
	{
		if (hipSuccess != hipMalloc(&pcuPrj, sizeof(float) * nu * nv * nview * nBatches * nChannels))
		{
			throw ("pcuPrj allocation failed");
		}
		if (hipSuccess != hipMalloc(&pcuImg, sizeof(float) * nx * ny * nz * nBatches * nChannels))
		{
			throw ("pcuImg allocation failed");
		}
		if (hipSuccess != hipMalloc(&pcuDetCenter, sizeof(float3) * nview))
		{
			throw ("pcuDetCenter allocation failed");
		}
		if (hipSuccess != hipMalloc(&pcuDetU, sizeof(float3) * nview))
		{
			throw ("pcuDetU allocation failed");
		}
		if (hipSuccess != hipMalloc(&pcuDetV, sizeof(float3) * nview))
		{
			throw ("pcuDetV allocation failed");
		}
		if (hipSuccess != hipMalloc(&pcuSrc, sizeof(float3) * nview))
		{
			throw ("pcuSrc allocation failed");
		}
	}
	catch (exception& e)
	{
		if (pcuPrj != NULL) hipFree(pcuPrj);
		if (pcuImg != NULL) hipFree(pcuImg);
		if (pcuDetCenter != NULL) hipFree(pcuDetCenter);
		if (pcuDetU != NULL) hipFree(pcuDetU);
		if (pcuDetV != NULL) hipFree(pcuDetV);
		if (pcuSrc != NULL) hipFree(pcuSrc);

		ostringstream oss;
		oss << "cSiddonParallelProjection() failed: " << e.what()
				<< " (" << hipGetErrorString(hipGetLastError()) << ")";
		cerr << oss.str() << endl;
		throw(oss.str().c_str());
	}

	hipMemcpy(pcuImg, img, sizeof(float) * nx * ny * nz * nBatches * nChannels, hipMemcpyHostToDevice);
	hipMemcpy(pcuDetCenter, detCenter, sizeof(float3) * nview, hipMemcpyHostToDevice);
	hipMemcpy(pcuDetU, detU, sizeof(float3) * nview, hipMemcpyHostToDevice);
	hipMemcpy(pcuDetV, detV, sizeof(float3) * nview, hipMemcpyHostToDevice);
	hipMemcpy(pcuSrc, src, sizeof(float3) * nview, hipMemcpyHostToDevice);
	hipMemset(pcuPrj, 0, sizeof(float) * nu * nview * nv * nBatches * nChannels);

	SiddonCone projector;
	projector.Setup(nBatches, nChannels, nx, ny, nz, dx, dy, dz, cx, cy, cz,
			nu, nview, nv, du, dv, off_u, off_v, 0, 0, 2);

	projector.ProjectionAbitrary(pcuImg, pcuPrj, pcuDetCenter, pcuDetU, pcuDetV, pcuSrc);
	hipMemcpy(prj, pcuPrj, sizeof(float) * nu * nv * nview * nBatches * nChannels, hipMemcpyDeviceToHost);

	hipFree(pcuPrj);
	hipFree(pcuImg);
	hipFree(pcuDetCenter);
	hipFree(pcuDetU);
	hipFree(pcuDetV);
	hipFree(pcuSrc);

}

extern "C" void cSiddonConeBackprojectionAbitrary(float* img, const float* prj,
		const float* detCenter, const float* detU, const float* detV, const float* src,
		int nBatches, int nChannels,
		int nx, int ny, int nz, float dx, float dy, float dz, float cx, float cy, float cz,
		int nu, int nview, int nv, float du, float dv, float off_u, float off_v)
{
	float* pcuPrj = NULL;
	float* pcuImg = NULL;
	float* pcuDetCenter = NULL;
	float* pcuDetU = NULL;
	float* pcuDetV = NULL;
	float* pcuSrc = NULL;

	try
	{
		if (hipSuccess != hipMalloc(&pcuPrj, sizeof(float) * nu * nv * nview * nBatches * nChannels))
		{
			throw ("pcuPrj allocation failed");
		}
		if (hipSuccess != hipMalloc(&pcuImg, sizeof(float) * nx * ny * nz * nBatches * nChannels))
		{
			throw ("pcuImg allocation failed");
		}
		if (hipSuccess != hipMalloc(&pcuDetCenter, sizeof(float3) * nview))
		{
			throw ("pcuDetCenter allocation failed");
		}
		if (hipSuccess != hipMalloc(&pcuDetU, sizeof(float3) * nview))
		{
			throw ("pcuDetU allocation failed");
		}
		if (hipSuccess != hipMalloc(&pcuDetV, sizeof(float3) * nview))
		{
			throw ("pcuDetV allocation failed");
		}
		if (hipSuccess != hipMalloc(&pcuSrc, sizeof(float3) * nview))
		{
			throw ("pcuSrc allocation failed");
		}
	}
	catch (exception& e)
	{
		if (pcuPrj != NULL) hipFree(pcuPrj);
		if (pcuImg != NULL) hipFree(pcuImg);
		if (pcuDetCenter != NULL) hipFree(pcuDetCenter);
		if (pcuDetU != NULL) hipFree(pcuDetU);
		if (pcuDetV != NULL) hipFree(pcuDetV);
		if (pcuSrc != NULL) hipFree(pcuSrc);

		ostringstream oss;
		oss << "cSiddonParallelProjection() failed: " << e.what()
				<< " (" << hipGetErrorString(hipGetLastError()) << ")";
		cerr << oss.str() << endl;
		throw(oss.str().c_str());
	}

	hipMemcpy(pcuPrj, prj, sizeof(float) * nu * nv * nview * nBatches * nChannels, hipMemcpyHostToDevice);
	hipMemcpy(pcuDetCenter, detCenter, sizeof(float3) * nview, hipMemcpyHostToDevice);
	hipMemcpy(pcuDetU, detU, sizeof(float3) * nview, hipMemcpyHostToDevice);
	hipMemcpy(pcuDetV, detV, sizeof(float3) * nview, hipMemcpyHostToDevice);
	hipMemcpy(pcuSrc, src, sizeof(float3) * nview, hipMemcpyHostToDevice);
	hipMemset(pcuImg, 0, sizeof(float) * nx * ny * nz * nBatches * nChannels);

	SiddonCone projector;
	projector.Setup(nBatches, nChannels, nx, ny, nz, dx, dy, dz, cx, cy, cz,
			nu, nview, nv, du, dv, off_u, off_v, 0, 0, 2);

	projector.BackprojectionAbitrary(pcuImg, pcuPrj, pcuDetCenter, pcuDetU, pcuDetV, pcuSrc);

	hipMemcpy(img, pcuImg, sizeof(float) * nx * ny * nz * nBatches * nChannels, hipMemcpyDeviceToHost);



	hipFree(pcuPrj);
	hipFree(pcuImg);
	hipFree(pcuDetCenter);
	hipFree(pcuDetU);
	hipFree(pcuDetV);
	hipFree(pcuSrc);

}

